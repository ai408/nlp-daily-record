#include "hip/hip_runtime.h" // CUDA����ʱAPI
#include <stdio.h> // ��׼�������

__global__ void helloFromGPU(void) // GPU�˺���
{
	printf("Hello World from GPU!\n"); //���Hello World from GPU!
}

int main(void) // ������
{
	// hello from cpu
	printf("Hello World from GPU!\n"); //CPU���������Hello World from CPU!

	helloFromGPU<<<1,10>>>(); // ����GPU�˺�����10���߳̿飬1��ʾÿ��grid��ֻ��1��block��10��ʾÿ��block����10���߳�
	hipDeviceReset(); // ���õ�ǰ�豸�ϵ�������Դ״̬����յ�ǰ�豸�ϵ������ڴ�

	return 0;
}